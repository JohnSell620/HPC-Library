#include "hip/hip_runtime.h"
/**
		gpu_densebench.cpp
		Description: Benchmarking of dense matrix-matrix multiplication on GPU.

		@author Johnny Sellers
		@version 1.1 06/11/2017
*/
#include <iostream>
#include <functional>
#include "Matrix.hpp"
#include "Timer.hpp"

__global__
void matmat_multiply(const Matrix *A, const Matrix *B, Matrix *C)
{
  multiply(A, B, C);
}


double runBench(int M, int N, int K, long numruns);

int main() {

	double clockspeed			= 3.4e9;
	double rate_h12		    = clockspeed*8/12;
	double rate_h20				= clockspeed*8/20;
	double rate_cbh12		  = clockspeed*8/12;
	double rate_cbh20			= clockspeed*8/20;

  double achieved_gpu = runBench(matmat_gpu);


	std::cout << "Routine Clock CPUID Loop-ops Scalar 2-wide 4-wide 4-wide-fma Achieved" << std::endl;

	std::cout << "hoisted " << clockspeed << " AVX2 12 " << rate_h12 << " " << 2*rate_h12 << " " << 4*rate_h12 << " " << 8*rate_h12 << " " << achieved_h << std::endl;
	std::cout << "hoisted " << clockspeed << " AVX2 20 " << rate_h20 << " " << 2*rate_h20 << " " << 4*rate_h20 << " " << 8*rate_h20 << " " << achieved_h << std::endl;
	std::cout << "copyblockhoisted " << clockspeed << " AVX2 12 " << rate_cbh12 << " " << 2*rate_cbh12 << " " << 4*rate_cbh12 << " " << 8*rate_cbh12 << " " << achieved_cbh << std::endl;
	std::cout << "copyblockhoisted " << clockspeed << " AVX2 20 " << rate_cbh20 << " " << 2*rate_cbh20 << " " << 4*rate_cbh20 << " " << 8*rate_cbh20 << " " << achieved_cbh << std::endl;

	return 0;

}


double runBench(int M, int N, int K, long numruns) {
  Matrix A(M, K), B(K, N), C(M, N);           // host copies
  Matrix *d_A(M,K), *d_B(K, N), *d_C(M, N);   // device copies

  // Allocate space for device copies
  hipMalloc((void **)&d_A, size_A);
  hipMalloc((void **)&d_A, size_B);
  hipMalloc((void **)&d_A, size_C);

  randomizeMatrix(A);
  randomizeMatrix(B);
  randomizeMatrix(C);

  // Copy inputs to device
  hipMemcpy(d_A, &A, size_A, hipMemcpyHostToDevice);
  hipMemcpy(d_B, &B, size_B, hipMemcpyHostToDevice);
  hipMemcpy(d_C, &C, size_C, hipMemcpyHostToDevice);

	double a = 0.0;
  for (long i = 8; i <= 4096/4; i *= 2) {
		long numruns = 8L*1048L/(i*i*i) + 2;
    // double t = bench(i, i, i, numruns, f);
    Timer T;
    T.start();
    for (int i = 0; i < numruns; ++i) {
      // f(A, B, C);
      matmat_multiply<<<1,1>>>(d_A, d_B, d_C);
    }
    T.stop();

    double t = T.elapsed();
    double flops_per_multiply = i*i*i;
		if (a < 2.0*1.e3*numruns*flops_per_multiply/t)
			a = 2.0*1.e3*numruns*flops_per_multiply/t;
  }

  // Copy results back to host
  hipMemcpy(&C, d_C, size_C, hipMemcpyDeviceToHost);

  // Cleanup
  hipFree(d_A); hipFree(d_B); hipFree(d_C);

	return a;
}
