#include "hip/hip_runtime.h"
/*
 * gpu_densebench.cpp
 * Description: Benchmarking of dense matrix-matrix
 * multiplication on GPU.
 * @author Johnny Sellers
 * @version 1.1 06/11/2017
 */
#include <iostream>
#include <functional>
#include <algorithm>
#include <random>
#include "Matrix.hpp"
#include "Timer.hpp"

#define BLOCK_SIZE 16

class GPUMatrix: public Matrix {
public:
  GPUMatrix(int M, int N):Matrix(M,N) {}
  GPUMatrix():Matrix() {}

  __host__ __device__ std::vector<double>& elements() {
    return arrayData;
  }
};

// Without shared memory
__global__
void MatMulKernel(double *A, double *B, double *C, int Awidth, int Bwidth) {
  double Cvalue = 0;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  for (int k = 0; k < Awidth; ++k)
    Cvalue += A[row*Awidth+k] * B[k*Bwidth+col];
  C[row*Bwidth+col] = Cvalue;
}

double runBench(int M, int N, int K) {
  // Host copies of GPUMatrix A, B, C
  GPUMatrix h_A(M, K), h_B(K, N), h_C(M, N);
  randomizeMatrix(h_A);
  randomizeMatrix(h_B);
  randomizeMatrix(h_C);

  // Copy host matrix elements to arrays for kernel
  double h_Aarr[M*K], h_Barr[K*N], h_Carr[M*N];
  std::copy(h_A.elements().begin(), h_A.elements().end(), h_Aarr);
  std::copy(h_B.elements().begin(), h_B.elements().end(), h_Barr);

  // Allocate space for device copies of A, B elements
  double *d_Aarr;
  size_t size = h_A.numRows()*h_A.numCols()*sizeof(double);
  hipMalloc((void **)&d_Aarr, size);
  hipMemcpy(d_Aarr, h_Aarr, size, hipMemcpyHostToDevice);
  double *d_Barr;
  size = K * N * sizeof(double);
  hipMalloc((void **)&d_Barr, size);
  hipMemcpy(d_Barr, h_Barr, size, hipMemcpyHostToDevice);
  double *d_Carr;
  size = M * N * sizeof(double);
  hipMalloc((void **)&d_Carr, size);

  // Block and grid dimensions for kernel
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid(h_B.numCols() / dimBlock.x, h_A.numRows() / dimBlock.y);

  Timer T;
  T.start();
  for (long i = 8; i <= 4096/4; i *= 2) {
    long numruns = 8L*1048L*1048L*16L/(i*i*i) + 2;
    for (int k = 0; k < numruns; ++k)
      MatMulKernel<<<dimGrid, dimBlock>>>(d_Aarr, d_Barr, d_Carr, K, N);
    T.stop();
  }
  T.stop();
  double t = T.elapsed();

  // Copy results back to host GPUMatrix
  hipMemcpy(h_Carr, d_Carr, size, hipMemcpyDeviceToHost);
  std::copy(h_Carr, h_Carr + size, std::back_inserter(h_C.elements()));

  // Test print 1-Norm of C
  Matrix& C = h_C;
  std::cout << "1-Norm of C: " << oneNorm(C) << std::endl;

  // Cleanup
  hipFree(d_Aarr);
  hipFree(d_Barr);
  hipFree(d_Carr);

	return t;
}

int main() {
  int dimA = 256, dimB = 16, dimC = 128;
  double t = runBench(dimA, dimB, dimC);
  std::cout << "Achived clockspeed: " << t << std::endl;
  return 0;
}
